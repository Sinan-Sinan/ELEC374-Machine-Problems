#include "hip/hip_runtime.h"
//Adam Bayley 20176309 19ahb Machine Problem 3 part 2

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <ctime>

#define S 16 //16x16, 256x256,... etc

int flag = 0; //flag for checking if matrices are =

//device matrix mult. calculates row and col of the grid / block and then
//flattens matrix before inserting values
__global__ void DeviceMatrixMultiplication(int *A, int *B, int *O, int size) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if (row < size && col < size) {
        float temp = 0;
        for (int i = 0; i < size; i++) {
            temp = temp + A[row*size + i] * B[i*size + col];
            O[row*size + col] = temp;
        }//close for i
    }//close if
}//close void devicematrix

void HostMatrixMultiplication(int *A, int *B, int *C, int size) {
    int offset1, offset2;
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            float temp = 0;
            for (int k = 0; k < size; k++) {
                offset1 = i*size + k;
                offset2 = k*size + j;

                temp = temp + A[offset1] * B[offset2];
            }//close for
            C[i*size + j] = temp;
        }//close for
    }//close for
}//close hostmatrix

int main() {

    //setup timer events
    hipEvent_t start1, stop1, start2, stop2;
    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&stop1);
    hipEventCreate(&stop2);

    //variable to keep track of time
    time_t t;

    float gpu_time1 = 0.0f;
    float gpu_time2 = 0.0f;

    //synchronize
    hipDeviceSynchronize();



    //seed the random values
    srand((unsigned)time(&t));

    //get the size of the matrix
    size_t hostSize = S*S*sizeof(int);

    //allocate host memory
    int* h_A = (int*)malloc(hostSize);
    int* h_B = (int*)malloc(hostSize);
    int* h_C = (int*)malloc(hostSize);
    int* h_P = (int*)malloc(hostSize);

    //initialize host matrix
    for (int i = 0; i < S; i++) {
        for (int j = 0; j < S; j++) {
            //get the 2 random values and assign
            int rand1 = rand() % 10;
            int rand2 = rand() % 10;
            *(h_A + i * S + j) = rand1;
            *(h_B + i * S + j) = rand2;
        }//close for j
    }//close for i

    //allocate device memory
    int* d_A;
    int* d_B;
    int* d_C;
    hipMalloc((void**)&d_A, hostSize);
    hipMalloc((void**)&d_B, hostSize);
    hipMalloc((void**)&d_C, hostSize);

    //time transfer of values and copy the memory
   // hipEventRecord(start1, 0); uncomment for P1
    hipMemcpy(d_A, h_A, hostSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, hostSize, hipMemcpyHostToDevice);
 //   hipEventRecord(stop1, 0); uncomment for P1
   // hipEventSynchronize(stop1); uncomment for P1

    //get the recorded time difference and print it out uncomment for P1
 //   hipEventElapsedTime(&gpu_time1, start1, stop1); uncomment for P1
 //   printf("Matrices transfer time: %0.2f \n", gpu_time1); uncomment for P1

    dim3 threadsPerBlock(16,16); //for p2: change this value according to the width needed
    dim3 numberOfBlocks(ceil(S / threadsPerBlock.x), ceil(S / threadsPerBlock.y), 1);


    hipEventRecord(start2, 0); //part 2
    DeviceMatrixMultiplication << <numberOfBlocks, threadsPerBlock >> >(d_A, d_B, d_C, S);
    hipEventRecord(stop2, 0); //part 2
    hipEventSynchronize(stop2); //part 2
    hipEventElapsedTime(&gpu_time2, start2, stop2); //part 2

    printf("for 16x16: \n");
    printf("number of blocks in x and y, respectively: %d, %d\n", (int)S/(int)16,(int)S/(int)16);
    printf("time taken : %0.2f ", gpu_time2);
    hipMemcpy(h_C, d_C, hostSize, hipMemcpyDeviceToHost);

    HostMatrixMultiplication(h_A, h_B, h_P, S);


    for (int x = 0; x < S; x++) {
        for (int y = 0; y < S; y++) {
            if (*(h_P + x * S + y) != *(h_C + x * S + y))
                flag = 1;
        }
    }
    if (flag == 0)
        printf("Test Passed.");
    else
        printf("Test Failed.");


}//close main






















